/*
 * Parallelized (Cuda) n-body solver for planets/stars in Euclid space
 *
 */


#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <sys/time.h>

#define BODY_COUNT 10000
#define SIM_STEPS 50            // Number of frames to simulate
#define G 6.674f * pow(10, -11) // Newton's universal const of gravity
#define DELTA_T 0.01f           // Time gap between two simulation frames (in simulation)

#define CHECK(call)                        \
  {                                        \
    gpuAssert((call), __FILE__, __LINE__); \
  }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort)
      exit(code);
  }
}

typedef struct
{
  float m, x, y, z, vx, vy, vz;
} Body;

int bye(double *tcalc)
{
  printf("Simulation took %lf seconds.\n", *tcalc);
  exit(0);
}

// Track CPU time
double cpuSecond()
{
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

__global__ void simulate(Body *bodies)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for (int step = 0; step < SIM_STEPS; step++)
    if (i < BODY_COUNT) // Ignore extra threads
    {
      float Fx = 0.0f;
      float Fy = 0.0f;
      float Fz = 0.0f;

      for (int j = 0; j < BODY_COUNT; j++)
      {
        if (i == j)
          continue;

        const float dx = bodies[j].x - bodies[i].x;
        const float dy = bodies[j].y - bodies[i].y;
        const float dz = bodies[j].z - bodies[i].z;
        const float dist = sqrt(dx * dx + dy * dy + dz * dz);
        const float dist_cubed = dist * dist * dist;

        // Calculate forces
        Fx += G * bodies[i].m * bodies[j].m / dist_cubed * dx;
        Fy += G * bodies[i].m * bodies[j].m / dist_cubed * dy;
        Fz += G * bodies[i].m * bodies[j].m / dist_cubed * dz;
      }

      // Assign velocities
      bodies[i].vx += DELTA_T * Fx;
      bodies[i].vy += DELTA_T * Fy;
      bodies[i].vz += DELTA_T * Fz;

      // Update coordinates
      bodies[i].x += bodies[i].vx * DELTA_T;
      bodies[i].y += bodies[i].vy * DELTA_T;
      bodies[i].z += bodies[i].vz * DELTA_T;
    }
}

int main(int argc, char *argv[])
{
  double tstart = 0.0, tstop = 0.0, tcalc = 0.0; // For timing
  Body *bodies;                                  // host copies of bodies
  Body *d_bodies;                                // device copies of bodies
  int size = BODY_COUNT * sizeof(Body);

  // Allocate memory for host copies of bodies
  bodies = (Body *)malloc(size);
  // Allocate memory for device copies of bodies
  hipMalloc((void **)&d_bodies, BODY_COUNT * sizeof(Body));

  char file_name[100], body_count[10];
  FILE *fp;

  sprintf(body_count, "%d", BODY_COUNT);
  strcpy(file_name, "/home/ubuntu/it17142038/assignment/dataset_");
  strcat(file_name, body_count);
  strcat(file_name, ".csv");
  fp = fopen(file_name, "r"); // read mode
  if (fp == NULL)
  {
    printf("Sorry, an error occured while reading input file.\n");
    return 0;
  }

  fscanf(fp, "%*[^\n]\n"); // Skip headings in input csv file

  // Assume csv with nx7 values (mass, coord_x, coord_y, coord_z, velocity_x, velocity_y, velocity_z)
  for (int i = 0; i < BODY_COUNT; i++)
    fscanf(fp, "%f,%f,%f,%f,%f,%f,%f", &bodies[i].m, &bodies[i].x, &bodies[i].y, &bodies[i].z, &bodies[i].vx, &bodies[i].vy, &bodies[i].vz);
  fclose(fp);

  CHECK(hipMemcpy(d_bodies, bodies, size, hipMemcpyHostToDevice)); // Copy bodies to device

  int block_size = 256; // Dim3 not necessary
  int num_blocks = (BODY_COUNT + block_size - 1) / block_size;

  // Timestamp
  tstart = cpuSecond();

  simulate<<<block_size, num_blocks>>>(d_bodies); // Launch simulate() kernal on GPU
  CHECK(hipDeviceSynchronize());                 // Wait for GPU to finish

  tstop = cpuSecond();

  CHECK(hipMemcpy(bodies, d_bodies, size, hipMemcpyDeviceToHost)); // Copy bodies to host

  printf("body0:\n m:%.7f\n x:%.7f\n y:%.7f\n z:%.7f\n vx:%.7f\n vy:%.7f\n vz:%.7f\n", bodies[0].m, bodies[0].x, bodies[0].y, bodies[0].z, bodies[0].vx, bodies[0].vy, bodies[0].vz);

  /* For debugging purposes */
  strcpy(file_name, "output_");
  strcat(file_name, "cuda_");
  strcat(file_name, body_count);
  strcat(file_name, ".csv");
  printf("Written all to %s\n", file_name);
  fp = fopen(file_name, "w"); // write mode
  if (fp == NULL)
  {
    printf("Sorry, an error occured while opening output file for writing.\n");
    return 0;
  }
  // Write headers for csv
  fprintf(fp, "mass,coord_x,coord_y,coord_z,velocity_x,velocity_y,velocity_z\n");
  // Write csv with nx7 values (mass, coord_x, coord_y, coord_z, velocity_x, velocity_y, velocity_z)
  for (int i = 0; i < BODY_COUNT; i++)
    fprintf(fp, "%.7f,%.7f,%.7f,%.7f,%.7f,%.7f,%.7f\n", bodies[i].m, bodies[i].x, bodies[i].y, bodies[i].z, bodies[i].vx, bodies[i].vy, bodies[i].vz);
  fclose(fp);

  free(bodies);       // Deallocate host memory
  hipFree(d_bodies); // Deallocate device memory

  printf("Simulated %d frames for %d bodies\n", SIM_STEPS, BODY_COUNT);
  tcalc = tstop - tstart;
  bye(&tcalc);

  return 0;
}
